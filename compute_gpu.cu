#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include "compute_gpu.cuh"

extern "C"
void compute_gpu(vector3* hVel, vector3* hPos, double* mass){
	// parallelize two dimensional array
	vector3* values;
	vector3** accels;
	hipMallocManaged(&values, sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	hipMallocManaged(&accels, sizeof(vector3*)*NUMENTITIES);
	for (int i=0; i<NUMENTITIES; i++) {
		accels[i] = &values[i*NUMENTITIES];
	}

	// thread calling kernel
	vector3* velTemp;
	vector3* posTemp;
	double* massTemp;
	dim3 dimBlock(10, 10);
	dim3 dimGrid(NUMENTITIES / dimBlock.x, NUMENTITIES / dimBlock.y);

	hipMallocManaged(&velTemp, sizeof(vector3)*NUMENTITIES);
	hipMallocManaged(&posTemp, sizeof(vector3)*NUMENTITIES);
	hipMallocManaged(&massTemp, sizeof(double)*NUMENTITIES);

	hipMemcpy(velTemp, hVel, sizeof(hVel), hipMemcpyHostToDevice);
	hipMemcpy(posTemp, hPos, sizeof(hPos), hipMemcpyHostToDevice);
	hipMemcpy(massTemp, mass, sizeof(mass), hipMemcpyHostToDevice);

	kernel<<<dimGrid, dimBlock>>>(accels, velTemp, posTemp, massTemp);
	second_kernel<<<dimGrid, dimBlock>>>(accels, velTemp, posTemp);

	hipMemcpy(hVel, velTemp, sizeof(velTemp), hipMemcpyDeviceToHost);
	hipMemcpy(hPos, posTemp, sizeof(posTemp), hipMemcpyDeviceToHost);
	hipMemcpy(mass, massTemp, sizeof(massTemp), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	// freeing memory
	hipFree(velTemp);
	hipFree(posTemp);
	hipFree(massTemp);
	hipFree(values);
	hipFree(accels);
}


__global__
void kernel(vector3** accels, vector3* hVel, vector3* hPos, double* mass){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y * threadIdx.y;

	if (i == j) {
		FILL_VECTOR(accels[i][j],0,0,0);
	} else {
		vector3 distance;
		for (int k=0; k<3; k++){
			distance[k] = hPos[i][k]-hPos[j][k];
		}
		double magnitude_sq = distance[0]*distance[0]+distance[1]+distance[2]*distance[2];
		double magnitude = sqrt(magnitude_sq);
		double accelmag = 1 * GRAV_CONSTANT * mass[j]/magnitude_sq;
		FILL_VECTOR(accels[i][j], accelmag * distance[0]/magnitude, accelmag*distance[1]/magnitude, accelmag*distance[2]/magnitude);
	}
}

__global__
void second_kernel(vector3** accels, vector3* hVel, vector3* hPos) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y * threadIdx.y;
	vector3 accel_sum = {0, 0, 0};
	int k;

	for (k=0;k<3;k++){
		accel_sum[k]+=accels[i][j][k];
	}

	for (k=0; k<3; k++){
		hVel[i][j]+=accel_sum[k]*INTERVAL;
		hPos[i][j]=hVel[i][k]*INTERVAL;
	}
}	
